#include "hip/hip_runtime.h"
#include "ColourHistogramGen.hpp"
#include <stdexcept>
#include <device_atomic_functions.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#include <iostream>

namespace chgen
{
	// Counts the unique values of each colour channel in the supplied image buffer
	// The format of the image should be RGB888
	__global__ void kImageAnalysis(int size, uint8_t* in,
		uint32_t* r, uint32_t* g, uint32_t* b)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;

		if (idx < size)
		{
			uint8_t val = in[idx];
			int c = idx % 3;

			/* increment the corresponding colour stats slot */
			switch (c)
			{
			case 0:
				atomicAdd(&r[val], 1);
			case 1:
				atomicAdd(&g[val], 1);
			case 2:
				atomicAdd(&b[val], 1);
			}
		}
	}

	// Plots the statistics of the colours to a image of 768x320
	// Output format is RGB888
	__global__ void kStatsToImage(uint32_t* r, uint32_t* g, uint32_t* b, uint32_t max, uint8_t* out)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;

		if (px < 768 && py < 320)
		{
			uint32_t* in;
			int c = px % 3;
			switch (c)
			{
			case 0:
				in = r;
				break;
			case 1:
				in = g;
				break;
			case 2:
				in = b;
				break;
			}

			/* calculate the y value of the top pixel of the current bar */
			float freq = static_cast<float>(in[px / 3]) / static_cast<float>(max);
			int top_y = static_cast<int>(320.0f - 320.0f * freq);

			/* draw */
			int index = (py * 768 + px) * 3;
			if (py >= top_y)
			{

				switch (c)
				{
				case 0:
					out[index] = 0xff;
					out[index + 1] = 0x00;
					out[index + 2] = 0x00;
					break;
				case 1:
					out[index] = 0x00;
					out[index + 1] = 0xff;
					out[index + 2] = 0x00;
					break;
				case 2:
					out[index] = 0x00;
					out[index + 1] = 0x00;
					out[index + 2] = 0xff;
					break;
				}
			}
			else
			{
				out[index] = 0x00;
				out[index + 1] = 0x00;
				out[index + 2] = 0x00;
			}
		}
	}
}

int chgen::CudaCount()
{
	int c = 0;
	hipGetDeviceCount(&c);
	return c;
}

chgen::ColourHistogramGen::ColourHistogramGen()
{
	/* allocate gpu memory */
	hipError_t err;
	err = hipMalloc(&gpu_stats_r, 256 * sizeof(uint32_t));
	if (err != hipSuccess)
		throw std::runtime_error("Failed to allocate device memory");

	err = hipMalloc(&gpu_stats_g, 256 * sizeof(uint32_t));
	if (err != hipSuccess)
	{
		hipFree(gpu_stats_r);
		throw std::runtime_error("Failed to allocate device memory");
	}

	err = hipMalloc(&gpu_stats_b, 256 * sizeof(uint32_t));
	if (err != hipSuccess)
	{
		hipFree(gpu_stats_r);
		hipFree(gpu_stats_g);
		throw std::runtime_error("Failed to allocate device memory");
	}

	/* zero the gpu memory */
	hipMemset(gpu_stats_r, 0, 256 * sizeof(uint32_t));
	hipMemset(gpu_stats_g, 0, 256 * sizeof(uint32_t));
	hipMemset(gpu_stats_b, 0, 256 * sizeof(uint32_t));
}

chgen::ColourHistogramGen::~ColourHistogramGen()
{
	/* deallocate gpu memory */
	hipFree(gpu_stats_r);
	hipFree(gpu_stats_g);
	hipFree(gpu_stats_b);
	if (hist_buf != nullptr)
		free(hist_buf);
}

void chgen::ColourHistogramGen::Analyse(chgen::Image& im)
{
	int size = im.width * im.height * 3;
	
	/* allocate gpu image buffer */
	uint8_t* gpu_imgbuf;
	hipError_t err = hipMalloc(&gpu_imgbuf, size);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to allocate device image buffer");

	/* copy image data to device */
	err = hipMemcpy(gpu_imgbuf, im.data, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to copy image to CUDA device buffer");

	/* call cuda kernel to do the statistics*/
	int block_size = 256;
	int grid_size = (size + block_size - 1) / block_size;

	kImageAnalysis<<<grid_size, block_size>>>(size, gpu_imgbuf, gpu_stats_r, gpu_stats_g, gpu_stats_b);
	hipDeviceSynchronize();

	/* free gpu image buffer */
	hipFree(gpu_imgbuf);
}

std::unique_ptr<struct chgen::ColourStats> chgen::ColourHistogramGen::GetColourStats()
{
	std::unique_ptr<struct ColourStats> cs = std::make_unique<struct ColourStats>();
	
	/* copy stats from gpu to cs heap */
	hipError_t err;
	err = hipMemcpy(cs->r, gpu_stats_r, 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to copy data from CUDA device");

	err = hipMemcpy(cs->g, gpu_stats_g, 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to copy data from CUDA device");

	err = hipMemcpy(cs->b, gpu_stats_b, 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to copy data from CUDA device");

	return cs;
}

std::unique_ptr<chgen::Image> chgen::ColourHistogramGen::GetHistogramImage()
{
	uint8_t* im_gpu;
	hipError_t err = hipMalloc(&im_gpu, 768 * 320 * 3 * sizeof(uint8_t));
	if (err != hipSuccess)
		throw std::runtime_error("Failed to allocate device histogram buffer");

	/* calculate the max value of the statistics arrays */
	thrust::device_ptr<uint32_t> th_r(gpu_stats_r);
	uint32_t max_r = *thrust::max_element(th_r, th_r + 255);

	thrust::device_ptr<uint32_t> th_g(gpu_stats_g);
	uint32_t max_g = *thrust::max_element(th_g, th_g + 255);

	thrust::device_ptr<uint32_t> th_b(gpu_stats_b);
	uint32_t max_b = *thrust::max_element(th_b, th_b + 255);

	uint32_t max = (max_g > max_r) ? max_g : max_r;
	max = (max > max_b) ? max : max_b;

	/* call kernel to generate histogram */
	dim3 block_dim(32, 32);
	dim3 grid_dim(24, 10);
	kStatsToImage<<<grid_dim, block_dim>>>(gpu_stats_r, gpu_stats_g, gpu_stats_b, max, im_gpu);

	/* allocate memory */
	hist_buf = (uint8_t*)malloc(768 * 320 * 3 * sizeof(uint8_t));

	/* copy image data */
	err = hipMemcpy(hist_buf, im_gpu, 768 * 320 * 3 * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		throw std::runtime_error("Failed to copy histogram image data");

	hipFree(im_gpu);

	std::unique_ptr<Image> im_ptr = std::make_unique<Image>();
	im_ptr->width = 768;
	im_ptr->height = 320;
	im_ptr->data = hist_buf;

	return im_ptr;
}
